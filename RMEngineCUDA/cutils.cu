
#include "cutils.cuh"

#include <iostream>

#include <hip/hip_runtime.h>



bool cutil::checkError(hipError_t error) {
  if (error != hipSuccess) {
    std::cout << "CUDA error: " << hipGetErrorString(error) << std::endl;
    return false;
  }
  return true;
};